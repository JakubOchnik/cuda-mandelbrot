#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include "../utilities/params.h"
#include<chrono>
#include<iostream>

class gpu_complex {
private:
    double im;
    double re;
public:
    __device__ gpu_complex(double x, double y) {
        re = x;
        im = y;
    }
    __device__ gpu_complex() {
        re = 0.0;
        im = 0.0;
    }
    __device__ double r_sq() {
        return re * re;
    }
    __device__ double im_sq() {
        return im * im;
    }
    __device__ double modulus() {
        return (r_sq() + im_sq());
    }
    __device__ gpu_complex operator*(const gpu_complex& a) {
        return gpu_complex(re * a.re - im * a.im, re * im + im * a.re);
    }
    __device__ gpu_complex operator+(const gpu_complex& a) {
        return gpu_complex(re + a.re, im + a.im);
    }
};

__device__ int checkIter(int x, int y, const bool color) {
    double s_x = x_min + x / (WIDTH -1.0) * (x_max - x_min);
    double s_y = y_min + y / (HEIGHT -1.0) * (y_max - y_min);
    gpu_complex num;
    gpu_complex C(s_x, s_y);
    int i = 0;
    for (; i < ITERATIONS; i++) {
        if (num.modulus() > 4)
            break;
        num = num * num + C;
    }
    if (!color) {
        if (i > 255)
            return 255;
        else
            return (unsigned char)i;
    }
    else {
        if (i >= ITERATIONS)
            return ITERATIONS;
        else
            return (unsigned char)i;
    }
}

__global__ void calculate(unsigned char* img)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int tid = (x + y * gridDim.x) * 3;
    int val = checkIter(x, y, 1);
    float ratio = (float)val * 1.0f / (float)ITERATIONS;
    unsigned char r, g, b;
    r = r_1 + ratio * (r_2 - r_1);
    g = g_1 + ratio * (g_2 - g_1);
    b = b_1 + ratio * (b_2 - b_1);
    img[tid] = b;//r_1 + val / 255.0 * (r_2 - r_1);
    img[tid + 1] = g;// g_1 + val / 255.0 * (g_2 - g_1);
    img[tid + 2] = r;//b_1 + val / 255.0 * (b_2 - b_1);
}

__global__ void calculate_mono(unsigned char* img)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int tid = x + y * gridDim.x;

    img[tid] = checkIter(x, y, 0);
}

void calculateMandelbrot(unsigned char* image) {
    dim3 blocksPerGrid(WIDTH, HEIGHT);
    unsigned char* dev_out;
    std::chrono::steady_clock::time_point start;
    size_t memSize;
    if (!color) 
        memSize = WIDTH * HEIGHT * sizeof(unsigned char);
    else 
        memSize = 3 * WIDTH * HEIGHT * sizeof(unsigned char);

    hipMalloc(&dev_out, memSize);
    hipMemcpy(dev_out, image, memSize, hipMemcpyHostToDevice);
    start = std::chrono::high_resolution_clock::now();
    if (!color) {
        calculate_mono << <blocksPerGrid, 1 >> > (dev_out);
    }
    else {
        calculate << <blocksPerGrid, 1 >> > (dev_out);
    }
    hipMemcpy(image, dev_out, memSize, hipMemcpyDeviceToHost);

    auto finish = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = finish - start;
    std::cout << "Elapsed time: " << elapsed.count() << " s\n";
    hipFree(dev_out);
}
